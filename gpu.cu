#include "hip/hip_runtime.h"
#include "common.h"
#include "assert.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

using namespace std;

#define NUM_THREADS 128

double bin_size; // bin_size = 2 * cutoff
int bin_Dim;  // number of stacked bins in each direction.
int num_bins; // = bin_Dim * bin_Dim
int* d_part_ids_by_bin;
int* d_bin_ids_prefix_sum;

// Put any static global variables here that you will use throughout the simulation.
int blks;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;

    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //  very simple short-range repulsive force
    double coef = (1 - cutoff / r) / r2 / mass;
    atomicAdd(&particle.ax, coef * dx);
    atomicAdd(&particle.ay, coef * dy);
}

/*
 * CUDA kernel that initializes acceleration components (ax, ay) of all particles to zero
 * This must be called before computing forces in each time step
 * 
 * Parameters:
 * particles: Array of particles in device memory
 * num_parts: Total number of particles
 * 
 * Note: No atomic operations needed here because:
 * 1. Each thread writes to its own unique particle
 * 2. No thread reads or writes to another thread's particle
 * 3. Each acceleration component is written only once
 */
__global__ void reset_forces_gpu(particle_t* particles, int num_parts) {
    // Calculate unique thread ID from block and thread indices
    // Each thread will handle resetting one particle's forces
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Return if this thread's ID exceeds the number of particles
    // (This happens because we round up the number of blocks)
    if (tid >= num_parts)
        return;

    // Reset both acceleration components to zero
    // This particle will accumulate forces from interactions in compute_forces_gpu
    particles[tid].ax = particles[tid].ay = 0;
}

// Returns the bin_id given the particle - GPU version
__device__ int get_bin_id_for_particle_gpu(particle_t* part, int bin_Dim, double bin_size){
    int bin_x = (int)(part -> x / bin_size);
    int bin_y = (int)(part -> y / bin_size);
    return bin_x + bin_y * bin_Dim;
}

__global__ void compute_forces_gpu(particle_t* d_parts, int num_parts, int bin_Dim, int* d_part_ids_by_bin, int* d_bin_ids_prefix_sum, double bin_size) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    int part1_id = d_part_ids_by_bin[tid];
    particle_t* particle_1_ptr = &d_parts[part1_id];
    int bin_id = get_bin_id_for_particle_gpu(&d_parts[part1_id], bin_Dim, bin_size);
    int bin_x_base = bin_id % bin_Dim;
    int bin_y_base = bin_id / bin_Dim;

    for(int bin_dy = -1; bin_dy <= 1; bin_dy++){
        for(int bin_dx = -1; bin_dx <= 1; bin_dx++){
            int bin_x = bin_x_base + bin_dx;
            int bin_y = bin_y_base + bin_dy;
            if(bin_x < 0 || bin_x >= bin_Dim || bin_y < 0 || bin_y >= bin_Dim)
                continue;
            int adj_bin_id = bin_x + bin_y * bin_Dim;
            // Iterate over all particles in the 'adj_bin_id'
            int n_parts_in_adj_bin = d_bin_ids_prefix_sum[adj_bin_id+1] - d_bin_ids_prefix_sum[adj_bin_id];
            for(int part2_local_id = 0; part2_local_id < n_parts_in_adj_bin; part2_local_id++){
                particle_t* particle_2_ptr = d_parts + d_part_ids_by_bin[d_bin_ids_prefix_sum[adj_bin_id] + part2_local_id];
                apply_force_gpu(*particle_1_ptr, *particle_2_ptr);
            }
        }
    }
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //  bounce from walls
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

/*
 * CUDA kernel that counts how many particles belong to each bin
 * Each thread processes one particle and atomically increments the count for that particle's bin
 * 
 * Parameters:
 * d_parts: Array of particles in device memory
 * num_parts: Total number of particles
 * d_bin_ids_prefix_sum: Array storing bin counts (will become prefix sum later)
 *                       d_bin_ids_prefix_sum[i+1] will store count of particles in bin i
 * bin_Dim: Number of bins in each row/column (grid is bin_Dim x bin_Dim)
 * bin_size: Physical size of each bin (= 2 * cutoff)
 */
__global__ void count_parts_in_bins(particle_t* d_parts, int num_parts, int* d_bin_ids_prefix_sum, int bin_Dim, double bin_size){
    // Calculate unique thread ID from block and thread indices
    // threadIdx.x: thread index within the block (0 to NUM_THREADS-1)
    // blockIdx.x: block index (0 to blks-1)
    // blockDim.x: number of threads per block (= NUM_THREADS)
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Return if this thread's ID exceeds the number of particles
    // (This happens because we round up the number of blocks)
    if (tid >= num_parts)
        return;

    // Each thread processes the particle with index = thread ID
    int& part_id = tid;

    // Calculate which bin this particle belongs to based on its position
    int bin_id = get_bin_id_for_particle_gpu(&d_parts[part_id], bin_Dim, bin_size);

    // Atomically increment the count for this bin
    // We use bin_id+1 because bin_ids_prefix_sum[0] needs to stay 0 for prefix sum
    // atomicAdd is needed because multiple threads might update the same bin count simultaneously
    atomicAdd(&d_bin_ids_prefix_sum[bin_id+1], 1);
}

/*
 * CUDA kernel that organizes particles into their respective bins
 * Each thread handles one particle and places its ID into the appropriate bin's section
 * 
 * Parameters:
 * d_parts: Array of particles in device memory
 * d_part_ids_by_bin: Output array where particle IDs will be stored, sorted by bin
 * d_bin_ids_prefix_sum: Array containing prefix sums of bin counts
 *                       d_bin_ids_prefix_sum[i] is the starting index for bin i's particles
 * num_parts: Total number of particles
 * bin_Dim: Number of bins in each row/column (grid is bin_Dim x bin_Dim)
 * bin_size: Physical size of each bin (= 2 * cutoff)
 */
__global__ void populate_bins(particle_t* d_parts, int* d_part_ids_by_bin, int* d_bin_ids_prefix_sum, int num_parts, int bin_Dim, double bin_size){
    // Calculate unique thread ID from block and thread indices
    // Each thread will handle one particle
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Return if this thread's ID exceeds the number of particles
    if (tid >= num_parts)
        return;

    // Each thread processes the particle with index = thread ID
    int& part_id = tid;

    // Calculate which bin this particle belongs to based on its position
    int bin_id = get_bin_id_for_particle_gpu(&d_parts[part_id], bin_Dim, bin_size);

    // Atomically claim a position in this bin's section of the array
    // atomicAdd returns the old value, which is our insertion position
    // We use bin_id+1 to maintain consistency with the prefix sum array
    int offset = atomicAdd(&d_bin_ids_prefix_sum[bin_id+1], 1);

    // Store the particle ID at the claimed position in the sorted array
    // This effectively groups particle IDs by their bin
    d_part_ids_by_bin[offset] = part_id;
}

void rebin_particles(particle_t* d_parts, int num_parts){
    // Step 1: Reset the prefix sum array to zeros
    // We need num_bins + 1 elements because prefix sum needs an extra element
    // Each bin will count its particles in bin_ids_prefix_sum[bin_id + 1]
    hipMemset(d_bin_ids_prefix_sum, 0, (num_bins+1)*sizeof(int));

    // Step 2: Count how many particles fall into each bin
    // Each thread handles one particle and atomically increments the count for that particle's bin
    // After this, bin_ids_prefix_sum[i+1] contains the count of particles in bin i
    count_parts_in_bins<<<blks, NUM_THREADS>>>(d_parts, num_parts, d_bin_ids_prefix_sum, bin_Dim, bin_size);

    // Step 3: Compute exclusive prefix sum of bin counts
    // Wrap raw device pointers in thrust device pointers
    thrust::device_ptr<int> dev_ptr_begin(d_bin_ids_prefix_sum);
    thrust::device_ptr<int> dev_ptr_end(d_bin_ids_prefix_sum + num_bins + 1);
    thrust::exclusive_scan(thrust::device, dev_ptr_begin, dev_ptr_end, dev_ptr_begin);

    // Step 4: Populate the bins array with particle IDs
    // Each thread handles one particle:
    // 1. Finds which bin the particle belongs to
    // 2. Atomically claims a position in that bin's section of the array
    // 3. Writes its particle ID to that position
    // After this, d_part_ids_by_bin contains particle IDs sorted by bin
    populate_bins<<<blks, NUM_THREADS>>>(d_parts, d_part_ids_by_bin, d_bin_ids_prefix_sum, num_parts, bin_Dim, bin_size);
}

/*
 * Allocates GPU memory for binning data structuresf
 * 
 * Parameters:
 * num_parts: Total number of particles (used for particle ID array size)
 * num_bins: Total number of bins (used for prefix sum array size)
 * 
 * Note: This function allocates:
 * 1. d_part_ids_by_bin: Array to store particle IDs grouped by bin
 * 2. d_bin_ids_prefix_sum: Array for bin counts and prefix sums
 */
void allocate_gpu_memory(int num_parts, int num_bins) {
    hipMalloc((void**)&d_part_ids_by_bin, num_parts * sizeof(int));
    hipMalloc((void**)&d_bin_ids_prefix_sum, (num_bins+1) * sizeof(int));
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here
    bin_size = 2 * cutoff;
    bin_Dim = ceil(size / bin_size);
    num_bins = bin_Dim * bin_Dim;
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
    
    // Allocate GPU memory for binning data structures
    allocate_gpu_memory(num_parts, num_bins);
}

void simulate_one_step(particle_t* d_parts, int num_parts, double size) {
    // parts live in GPU memory
    // Rewrite this function
    rebin_particles(d_parts, num_parts);
    reset_forces_gpu<<<blks, NUM_THREADS>>>(d_parts, num_parts);
    compute_forces_gpu<<<blks, NUM_THREADS>>>(d_parts, num_parts, bin_Dim, d_part_ids_by_bin, d_bin_ids_prefix_sum, bin_size);
    move_gpu<<<blks, NUM_THREADS>>>(d_parts, num_parts, size);
}