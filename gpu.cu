#include "hip/hip_runtime.h"
#include "common.h"
#include "assert.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

using namespace std;

#define NUM_THREADS 128

double bin_size; // bin_size = 2 * cutoff
int bin_Dim;  // number of stacked bins in each direction.
int num_bins; // = bin_Dim * bin_Dim
int* d_part_ids_by_bin;
int* d_bin_ids_prefix_sum;

// Put any static global variables here that you will use throughout the simulation.
int blks;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;

    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //  very simple short-range repulsive force
    double coef = (1 - cutoff / r) / r2 / mass;
    atomicAdd(&particle.ax, coef * dx);
    atomicAdd(&particle.ay, coef * dy);
}

/*
 * CUDA kernel that initializes acceleration components (ax, ay) of all particles to zero
 * This must be called before computing forces in each time step
 * 
 * Parameters:
 * particles: Array of particles in device memory
 * num_parts: Total number of particles
 * 
 * Note: No atomic operations needed here because:
 * 1. Each thread writes to its own unique particle
 * 2. No thread reads or writes to another thread's particle
 * 3. Each acceleration component is written only once
 */
__global__ void reset_forces_gpu(particle_t* particles, int num_parts) {
    // Calculate unique thread ID from block and thread indices
    // Each thread will handle resetting one particle's forces
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Return if this thread's ID exceeds the number of particles
    // (This happens because we round up the number of blocks)
    if (tid >= num_parts)
        return;

    // Reset both acceleration components to zero
    // This particle will accumulate forces from interactions in compute_forces_gpu
    particles[tid].ax = particles[tid].ay = 0;
}

// Returns the bin_id given the particle - GPU version
__device__ int get_bin_id_for_particle_gpu(particle_t* part, int bin_Dim, double bin_size){
    int bin_x = (int)(part -> x / bin_size);
    int bin_y = (int)(part -> y / bin_size);
    return bin_x + bin_y * bin_Dim;
}

__global__ void compute_forces_gpu(particle_t* d_parts, int num_parts, int bin_Dim, int* d_part_ids_by_bin, int* d_bin_ids_prefix_sum, double bin_size) {
    // Get thread (particle) ID based on global thread indexing
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;
    
    // Declare shared memory for particles in the current block
    extern __shared__ particle_t shared_parts[];
    
    // Get the actual particle ID from the binned array
    int part1_id = d_part_ids_by_bin[tid];
    particle_t& my_particle = d_parts[part1_id];
    
    // Calculate bin coordinates directly to match your existing approach
    int bin_x_base = (int)(my_particle.x / bin_size);
    int bin_y_base = (int)(my_particle.y / bin_size);
    
    // Local accumulators to reduce atomic operations
    double ax_local = 0.0;
    double ay_local = 0.0;
    
    // Examine all neighboring bins (including the particle's own bin)
    for(int bin_dy = -1; bin_dy <= 1; bin_dy++){
        for(int bin_dx = -1; bin_dx <= 1; bin_dx++){
            int bin_x = bin_x_base + bin_dx;
            int bin_y = bin_y_base + bin_dy;
            
            // Skip bins that are out of bounds
            if(bin_x < 0 || bin_x >= bin_Dim || bin_y < 0 || bin_y >= bin_Dim)
                continue;
            
            int adj_bin_id = bin_x + bin_y * bin_Dim;
            
            // Get range of particles in this bin using the prefix sum
            int bin_start = d_bin_ids_prefix_sum[adj_bin_id];
            int bin_end = d_bin_ids_prefix_sum[adj_bin_id+1];
            int n_parts_in_adj_bin = bin_end - bin_start;
            
            // Skip empty bins
            if (n_parts_in_adj_bin == 0)
                continue;
            
            // Process particles in chunks that fit in shared memory
            for (int chunk_start = 0; chunk_start < n_parts_in_adj_bin; chunk_start += blockDim.x) {
                int chunk_size = min(blockDim.x, n_parts_in_adj_bin - chunk_start);
                
                // Collaboratively load particles into shared memory
                if (threadIdx.x < chunk_size) {
                    int particle_index = bin_start + chunk_start + threadIdx.x;
                    int part_idx = d_part_ids_by_bin[particle_index];
                    shared_parts[threadIdx.x] = d_parts[part_idx];
                }
                
                // Ensure all threads have loaded their particles before proceeding
                __syncthreads();
                
                // Process all particles in this chunk
                for (int j = 0; j < chunk_size; j++) {
                    // Get particle ID for self-interaction check
                    int neighbor_idx = bin_start + chunk_start + j;
                    int neighbor_part_id = d_part_ids_by_bin[neighbor_idx];
                    
                    // Skip self-interaction
                    if (neighbor_part_id == part1_id)
                        continue;
                        
                    // Access particle from shared memory
                    particle_t& neighbor = shared_parts[j];
                    
                    // Apply force calculation - identical to original algorithm
                    double dx = neighbor.x - my_particle.x;
                    double dy = neighbor.y - my_particle.y;
                    double r2 = dx * dx + dy * dy;
                    
                    if (r2 > cutoff * cutoff)
                        continue;
                    
                    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
                    double r = sqrt(r2);
                    
                    // Same force calculation as original
                    double coef = (1 - cutoff / r) / r2 / mass;
                    ax_local += coef * dx;
                    ay_local += coef * dy;
                }
                
                // Ensure all threads are done with shared memory before next iteration
                __syncthreads();
            }
        }
    }
    
    // Update the particle's acceleration with locally accumulated values
    atomicAdd(&d_parts[part1_id].ax, ax_local);
    atomicAdd(&d_parts[part1_id].ay, ay_local);
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //  bounce from walls
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

/*
 * CUDA kernel that counts how many particles belong to each bin
 * Each thread processes one particle and atomically increments the count for that particle's bin
 * 
 * Parameters:
 * d_parts: Array of particles in device memory
 * num_parts: Total number of particles
 * d_bin_ids_prefix_sum: Array storing bin counts (will become prefix sum later)
 *                       d_bin_ids_prefix_sum[i+1] will store count of particles in bin i
 * bin_Dim: Number of bins in each row/column (grid is bin_Dim x bin_Dim)
 * bin_size: Physical size of each bin (= 2 * cutoff)
 */
__global__ void count_parts_in_bins(particle_t* d_parts, int num_parts, int* d_bin_ids_prefix_sum, int bin_Dim, double bin_size){
    // Calculate unique thread ID from block and thread indices
    // threadIdx.x: thread index within the block (0 to NUM_THREADS-1)
    // blockIdx.x: block index (0 to blks-1)
    // blockDim.x: number of threads per block (= NUM_THREADS)
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Return if this thread's ID exceeds the number of particles
    // (This happens because we round up the number of blocks)
    if (tid >= num_parts)
        return;

    // Each thread processes the particle with index = thread ID
    int& part_id = tid;

    // Calculate which bin this particle belongs to based on its position
    int bin_id = get_bin_id_for_particle_gpu(&d_parts[part_id], bin_Dim, bin_size);

    // Atomically increment the count for this bin
    // We use bin_id+1 because bin_ids_prefix_sum[0] needs to stay 0 for prefix sum
    // atomicAdd is needed because multiple threads might update the same bin count simultaneously
    atomicAdd(&d_bin_ids_prefix_sum[bin_id+1], 1);
}

/*
 * CUDA kernel that organizes particles into their respective bins
 * Each thread handles one particle and places its ID into the appropriate bin's section
 * 
 * Parameters:
 * d_parts: Array of particles in device memory
 * d_part_ids_by_bin: Output array where particle IDs will be stored, sorted by bin
 * d_bin_ids_prefix_sum: Array containing prefix sums of bin counts
 *                       d_bin_ids_prefix_sum[i] is the starting index for bin i's particles
 * num_parts: Total number of particles
 * bin_Dim: Number of bins in each row/column (grid is bin_Dim x bin_Dim)
 * bin_size: Physical size of each bin (= 2 * cutoff)
 */
__global__ void populate_bins(particle_t* d_parts, int* d_part_ids_by_bin, int* d_bin_ids_prefix_sum, int num_parts, int bin_Dim, double bin_size){
    // Calculate unique thread ID from block and thread indices
    // Each thread will handle one particle
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Return if this thread's ID exceeds the number of particles
    if (tid >= num_parts)
        return;

    // Each thread processes the particle with index = thread ID
    int& part_id = tid;

    // Calculate which bin this particle belongs to based on its position
    int bin_id = get_bin_id_for_particle_gpu(&d_parts[part_id], bin_Dim, bin_size);

    // Atomically claim a position in this bin's section of the array
    // atomicAdd returns the old value, which is our insertion position
    // We use bin_id+1 to maintain consistency with the prefix sum array
    int offset = atomicAdd(&d_bin_ids_prefix_sum[bin_id+1], 1);

    // Store the particle ID at the claimed position in the sorted array
    // This effectively groups particle IDs by their bin
    d_part_ids_by_bin[offset] = part_id;
}

void rebin_particles(particle_t* d_parts, int num_parts){
    // Step 1: Reset the prefix sum array to zeros
    // We need num_bins + 1 elements because prefix sum needs an extra element
    // Each bin will count its particles in bin_ids_prefix_sum[bin_id + 1]
    hipMemset(d_bin_ids_prefix_sum, 0, (num_bins+1)*sizeof(int));

    // Step 2: Count how many particles fall into each bin
    // Each thread handles one particle and atomically increments the count for that particle's bin
    // After this, bin_ids_prefix_sum[i+1] contains the count of particles in bin i
    count_parts_in_bins<<<blks, NUM_THREADS>>>(d_parts, num_parts, d_bin_ids_prefix_sum, bin_Dim, bin_size);

    // Step 3: Compute exclusive prefix sum of bin counts
    // Wrap raw device pointers in thrust device pointers
    thrust::device_ptr<int> dev_ptr_begin(d_bin_ids_prefix_sum);
    thrust::device_ptr<int> dev_ptr_end(d_bin_ids_prefix_sum + num_bins + 1);
    thrust::exclusive_scan(thrust::device, dev_ptr_begin, dev_ptr_end, dev_ptr_begin);

    // Step 4: Populate the bins array with particle IDs
    // Each thread handles one particle:
    // 1. Finds which bin the particle belongs to
    // 2. Atomically claims a position in that bin's section of the array
    // 3. Writes its particle ID to that position
    // After this, d_part_ids_by_bin contains particle IDs sorted by bin
    populate_bins<<<blks, NUM_THREADS>>>(d_parts, d_part_ids_by_bin, d_bin_ids_prefix_sum, num_parts, bin_Dim, bin_size);
}

/*
 * Allocates GPU memory for binning data structuresf
 * 
 * Parameters:
 * num_parts: Total number of particles (used for particle ID array size)
 * num_bins: Total number of bins (used for prefix sum array size)
 * 
 * Note: This function allocates:
 * 1. d_part_ids_by_bin: Array to store particle IDs grouped by bin
 * 2. d_bin_ids_prefix_sum: Array for bin counts and prefix sums
 */
void allocate_gpu_memory(int num_parts, int num_bins) {
    hipMalloc((void**)&d_part_ids_by_bin, num_parts * sizeof(int));
    hipMalloc((void**)&d_bin_ids_prefix_sum, (num_bins+1) * sizeof(int));
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here
    bin_size = 2 * cutoff;
    bin_Dim = ceil(size / bin_size);
    num_bins = bin_Dim * bin_Dim;
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
    
    // Allocate GPU memory for binning data structures
    allocate_gpu_memory(num_parts, num_bins);
}

void simulate_one_step(particle_t* d_parts, int num_parts, double size) {
    // Rebin particles
    rebin_particles(d_parts, num_parts);
    
    // Reset forces
    reset_forces_gpu<<<blks, NUM_THREADS>>>(d_parts, num_parts);
    
    // Compute forces with shared memory
    size_t shared_mem_size = NUM_THREADS * sizeof(particle_t);
    compute_forces_gpu<<<blks, NUM_THREADS, shared_mem_size>>>(d_parts, num_parts, bin_Dim, d_part_ids_by_bin, d_bin_ids_prefix_sum, bin_size);
    
    // Move particles
    move_gpu<<<blks, NUM_THREADS>>>(d_parts, num_parts, size);
}